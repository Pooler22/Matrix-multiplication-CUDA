#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define RAND_MAX 100
#define TILE_WIDTH 2

int size;

void matrixMultiplicationCPU(int* inputA, int* inputB, int* output)
{
	int i, j, k;
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			for (k = 0; k < size; k++)
				output[i*size + j] = output[i*size + j] + inputA[i*size + k] * inputB[j*size + j];
}

__global__ void matrixMultiplicationGPU(int *inputA, int *inputB, int *output, int size)
{
	int i, sum ;
	int columns = threadIdx.x + blockDim.x * blockIdx.x;
	int rows = threadIdx.y + blockDim.y * blockIdx.y;

	if (columns < size && rows < size)
	{
		sum = 0;
		for (i = 0; i < size; i++)
			sum += inputA[rows * size + i] * inputB[i * size + columns];
		output[rows * size + columns] = sum;
	}
}

__global__ void matrixMultiplicationGPUSharedMemeory(int *inputA, int *inputB, int *output, int size)
{
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
	int tIdX = threadIdx.x;
	int tIdY = threadIdx.y;
	int row = blockIdx.y * TILE_WIDTH + tIdY;
	int column = blockIdx.x * TILE_WIDTH + tIdX;
	int i, j, sum = 0;;
	
	for (i = 0; i < size / TILE_WIDTH; i++)
	{
		
		Mds[tIdY][tIdX] = inputA[row * size + (i * TILE_WIDTH + tIdX)];
		Nds[tIdY][tIdX] = inputB[(i * TILE_WIDTH + tIdY) * size + column];
		__syncthreads();
		for (j = 0; j < TILE_WIDTH; j++)
			sum += Mds[tIdY][j] * Nds[j][tIdX];
		__syncthreads();
	}
	output[row * size + column] = sum;
}

int* generateArray(int count)
{
	int *array;
	srand(time(NULL));
	array = (int*)malloc(count * sizeof(int));
	for (int i = 0; i < count; i++)
		(array)[i] = rand() % RAND_MAX;
	return array;
}

void saveToFile(int* array, char* name, int size)
{
	int i, j;
	FILE *file = fopen(name, "a");
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
			fprintf(file, "%d\t", array[i * size + j]);
		fprintf(file, "\n");
	}
	fclose(file);
}

void CPU(int* inputA, int* inputB, int* output, FILE *fileTime)
{
	LARGE_INTEGER frequency, start, end;

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);
	
	matrixMultiplicationCPU(inputA, inputB, output);
	
	QueryPerformanceCounter(&end);

	saveToFile(output, "outMatrixCPU.txt", size);
	fprintf(fileTime, "%f\t", ((double)(end.QuadPart - start.QuadPart) / frequency.QuadPart) * 1000);
}

void GPU(int* inputA, int* inputB, int* output, FILE *fileTime)
{
	int *dev_inputA, *dev_inputB, *dev_output;
	float time;
	LARGE_INTEGER frequency, start, end;

	hipMalloc((void **)&dev_inputA, size * size * sizeof(int));
	hipMalloc((void **)&dev_inputB, size * size * sizeof(int));
	hipMalloc((void **)&dev_output, size * size * sizeof(int));

	hipMemcpy(dev_inputA, inputA, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_inputB, inputB, size * size * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(size, size);
	dim3 dimGrid(1, 1);
	//dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(size/dimBlock.y));

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);
	matrixMultiplicationGPU << <dimGrid, dimBlock >> >(dev_inputA, dev_inputB, dev_output, size);
	hipDeviceSynchronize();
	hipMemcpy(output, dev_output, size * size * sizeof(int), hipMemcpyDeviceToHost);
	QueryPerformanceCounter(&end);
	
	hipError_t cudaStatus;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "error: %s\n", hipGetErrorString(cudaStatus));
	}
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			printf("%d\t", output[i * size + j]);
		printf("\n");
	}

	saveToFile(output, "outMatrixGPU.txt", size);
	fprintf(fileTime, "%f\t", ((double)(end.QuadPart - start.QuadPart) / frequency.QuadPart) * 1000);

	hipFree(dev_inputA);
	hipFree(dev_inputB);
	hipFree(dev_output);
}


void GPUSM(int* inputA, int* inputB, int* output, FILE *fileTime)
{
	int *dev_inputA, *dev_inputB, *dev_output;
	float time;
	LARGE_INTEGER frequency, start, end;

	hipMalloc((void **)&dev_inputA, size * size * sizeof(int));
	hipMalloc((void **)&dev_inputB, size * size * sizeof(int));
	hipMalloc((void **)&dev_output, size * size * sizeof(int));

	hipMemcpy(dev_inputA, inputA, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_inputB, inputB, size * size * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGridSM(size / TILE_WIDTH, size / TILE_WIDTH);
	dim3 dimBlockSM(TILE_WIDTH, TILE_WIDTH);

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);
	matrixMultiplicationGPUSharedMemeory << <dimGridSM, dimBlockSM >> >(dev_inputA, dev_inputB, output, size);
	hipDeviceSynchronize();
	hipMemcpy(output, dev_output, size * size * sizeof(int), hipMemcpyDeviceToHost);
	QueryPerformanceCounter(&end);

	saveToFile(output, "outMatrixGPUSH.txt", size);
	fprintf(fileTime, "%f\t", ((double)(end.QuadPart - start.QuadPart) / frequency.QuadPart) * 1000);

	hipFree(dev_inputA);
	hipFree(dev_inputB);
	hipFree(dev_output);
}

void init(int** inputA, int** inputB, int** output)
{
	*inputA = generateArray(size * size);
	*inputB = generateArray(size * size);
	*output = (int*)malloc(size * size * sizeof(int));
	for (int i = 0; i < size * size; i++)
		(*output)[i] = 0;
}

int main() {
	int *inputA, *inputB, *output;
	FILE *fileTime = fopen("outTime.txt", "a");
	for (int i = 10; i < 12; i+= 16)
	{
		size = i;
		init(&inputA, &inputB, &output);
		
		CPU(inputA, inputB, output, fileTime);

		for (int k = 0; k < size * size; k++)
			output[k] = 0;

		GPU(inputA, inputB, output, fileTime);

		for (int k = 0; k < size * size; k++)
			output[k] = 0;

		GPUSM(inputA, inputB, output, fileTime);

		free(inputA);
		free(inputB);
		free(output);
	}
	

	system("PAUSE");
	return 0;
}