#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>


#define N 100
#define RAND_MAX 100
#define TILE_WIDTH 2

void matrixMultiplicationCPU(float* inputA, float* inputB, float* output)
{
	int i, j, k;

	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++)
			for (k = 0; k < N; k++)
				output[i * N + j] = output[i * N + j] + inputA[i * N + k] * inputB[k * N + j];
}

__global__ void matrixMultiplicationGPU(float *inputA, float *inputB, float *output, int size)
{
	int i, sum = 0;
	int columns = threadIdx.x + blockDim.x * blockIdx.x;
	int rows = threadIdx.y + blockDim.y * blockIdx.y;

	if (columns < size && rows < size)
	{
		for (i = 0; i < size; i++)
			sum += inputA[rows * size + i] * inputB[i * size + columns];
		output[rows * size + columns] = sum;
	}
}

__global__ void matrixMultiplicationGPUSharedMemeory(float *inputA, float *inputB, float *output, int size)
{
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
	int tIdX = threadIdx.x;
	int tIdY = threadIdx.y;
	int row = blockIdx.y * TILE_WIDTH + tIdY;
	int column = blockIdx.x * TILE_WIDTH + tIdX;
	int i, j, sum = 0;
	
	for (i = 0; i < size / TILE_WIDTH; i++)
	{
		Mds[tIdY][tIdX] = inputA[row * size + (i * TILE_WIDTH + tIdX)];
		Nds[tIdY][tIdX] = inputB[(i * TILE_WIDTH + tIdY) * size + column];
		__syncthreads();
		for (j = 0; j < TILE_WIDTH; j++)
			sum += Mds[tIdY][j] * Nds[j][tIdX];
		__syncthreads();
	}
	output[row * size + column] = sum;
}

float* generateArray(int count)
{
	float *array;
	srand(time(NULL));
	array = (float*)malloc(count * sizeof(float));
	for (int i = 0; i < count; i++)
		(array)[i] = rand() % RAND_MAX;
	return array;
}

void saveToFile(float* array, char* name, int size)
{
	int i, j;
	FILE *file = fopen(name, "a");
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
			fprintf(file, "%d\t", array[i * size + j]);
		fprintf(file, "\n");
	}
	fclose(file);
}


int main() {
	float *inputA, *inputB, *output, *dev_inputA, *dev_inputB, *dev_output, i, j, size = N * N * sizeof(float);
	float time;
	hipEvent_t start, stop, startSM, stopSM;
	LARGE_INTEGER frequency, startCPU, endCPU;
	FILE *fileTime = fopen("outTime.txt", "a");
	
	//prepare array
	inputA = generateArray(N * N);
	inputB = generateArray(N * N);
	output = (float*) malloc(size);
	for (int i = 0; i < N * N; i++)
		output[i] = 0;
	
	//CPU
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&startCPU);
	//CPU calculations
	matrixMultiplicationCPU(inputA, inputB, output);
	QueryPerformanceCounter(&endCPU);
	
	//save to file
	saveToFile(output, "outMatrixCPU.txt", N);
	fprintf(fileTime, "CPU time %f ms\n", ((double)(endCPU.QuadPart - startCPU.QuadPart) / frequency.QuadPart) * 1000);

	//GPU
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMalloc((void **)&dev_inputA, size);
	hipMalloc((void **)&dev_inputB, size);
	hipMalloc((void **)&dev_output, size);

	hipMemcpy(dev_inputA, inputA, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_inputB, inputB, size, hipMemcpyHostToDevice);

	dim3 dimBlock(N, N);
	dim3 dimGrid(1, 1);
	//dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

	hipEventRecord(start, 0);
	//GPU calculations
	matrixMultiplicationGPU <<<dimGrid, dimBlock >>>(dev_inputA, dev_inputB, output, N);
	
	hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	saveToFile(output, "outMatrixGPU.txt", N);
	fprintf(fileTime, "GPU time %g ms\n", time);

	hipFree(dev_inputA);
	hipFree(dev_inputB);
	hipFree(dev_output);

	//GPU + SM
	hipEventCreate(&startSM);
	hipEventCreate(&stopSM);
	hipEventRecord(startSM, 0);
	hipMalloc((void **)&dev_inputA, size);
	hipMalloc((void **)&dev_inputB, size);
	hipMalloc((void **)&dev_output, size);

	hipMemcpy(dev_inputA, inputA, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_inputB, inputB, size, hipMemcpyHostToDevice);

	dim3 dimGridSM(N / TILE_WIDTH, N / TILE_WIDTH);
	dim3 dimBlockSM(TILE_WIDTH, TILE_WIDTH);

	hipEventRecord(startSM, 0);
	
	//GPU calculations
	matrixMultiplicationGPUSharedMemeory <<<dimGridSM, dimBlockSM >>>(dev_inputA, dev_inputB, output, N);
	
	hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);
	hipEventRecord(stopSM, 0);
	hipEventSynchronize(stopSM);
	hipEventElapsedTime(&time, startSM, stopSM);

	saveToFile(output, "outMatrixGPUSM.txt", N);
	fprintf(fileTime, "GPU SM time %g ms\n", time);

	hipFree(dev_inputA);
	hipFree(dev_inputB);
	hipFree(dev_output);

	system("PAUSE");
	return 0;
}